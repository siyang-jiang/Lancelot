// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT license.

#include "util/hash.h"
#include <cstdint>
#include <gtest/gtest.h>

using namespace std;
using namespace cahel::util;

namespace CAHELtest
{
    namespace util
    {
        namespace
        {
            void hash(uint64_t value, HashFunction::hash_block_type &destination)
            {
                HashFunction::hash(&value, 1, destination);
            }
        } // namespace

        TEST(HashTest, Hash)
        {
            uint64_t input[3]{ 0, 0, 0 };
            HashFunction::hash_block_type hash1, hash2;
            hash(0, hash1);

            HashFunction::hash(input, 0, hash2);
            ASSERT_TRUE(hash1 != hash2);

            HashFunction::hash(input, 1, hash2);
            ASSERT_TRUE(hash1 == hash2);

            HashFunction::hash(input, 2, hash2);
            ASSERT_TRUE(hash1 != hash2);

            hash(0x123456, hash1);
            hash(0x023456, hash2);
            ASSERT_TRUE(hash1 != hash2);

            input[0] = 0x123456;
            input[1] = 1;
            hash(0x123456, hash1);
            HashFunction::hash(input, 2, hash2);
            ASSERT_TRUE(hash1 != hash2);
        }
    } // namespace util
} // namespace CAHELtest
